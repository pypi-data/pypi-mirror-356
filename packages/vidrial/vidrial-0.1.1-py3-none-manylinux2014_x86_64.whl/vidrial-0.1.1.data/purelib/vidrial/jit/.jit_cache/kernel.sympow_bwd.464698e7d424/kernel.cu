
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cute/tensor.hpp"
#include "cutlass/cutlass.h"
#include "sympow_mosaic.cuh"
#include "sympow_bwd/kernel.cuh"

using namespace cute;
using namespace mosaic;

extern "C" void launch(void* __raw_X, void* __raw_Zgrad, void* __raw_Xgrad) {
    using T = bfloat16_t;
    auto X = reinterpret_cast<T*>(__raw_X);
    auto Zgrad = reinterpret_cast<T*>(__raw_Zgrad);
    auto Xgrad = reinterpret_cast<T*>(__raw_Xgrad);
    using GXSlab = Layout<Shape<Int<64>, Shape<Int<16>, Int<4096>>>, Stride<Int<4096>, Stride<Int<262144>, Int<1>>>>;
    using GZSlab = Layout<Shape<Shape<Shape<Int<8>, Int<8>>, Int<36>>, Shape<Int<16>, Int<4096>>>, Stride<Stride<Stride<Int<4096>, Int<32768>>, Int<262144>>, Stride<Int<9437184>, Int<1>>>>;
    constexpr int p = 2;
    constexpr int d_tile = 8;
    constexpr int b_tile = 4;
    using ZFrgShape = Shape<Shape<Int<4>, Int<4>>, Int<2>>;
    constexpr bool duplicate_correction = true;
    using XSlabShape = Shape<Int<64>, Shape<Int<16>, Int<4096>>>;
    using ZSlabShape = decltype(sympow_shape<p, d_tile>(XSlabShape{}));
    using XTileShape = Shape<Int<d_tile>, Int<b_tile>>; 
    using ZTileShape = decltype(tpow_shape<p>(XTileShape{}));
    using SZTileLayout = Layout<Shape<Shape<Int<8>, Int<8>>, Int<4>>, Stride<Stride<Int<1>, Int<8>>, Int<64>>>;
    using ZFrgThr = decltype(zipped_divide(Layout<ZTileShape>{}, ZFrgShape{}));
    auto mos = SympowMosaic<T, p, XSlabShape, XTileShape, ZFrgThr, GZSlab, GXSlab, SZTileLayout>{};
    launch_tiled_sympow_bwd_kernel<duplicate_correction>(mos, X, Zgrad, Xgrad);
}
