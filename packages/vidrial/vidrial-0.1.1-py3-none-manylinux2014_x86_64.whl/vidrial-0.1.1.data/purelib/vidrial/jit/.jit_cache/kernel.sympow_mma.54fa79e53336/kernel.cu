#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cute/tensor.hpp"
#include "sympow_mma/kernel.cuh"

using namespace cute;
using namespace mosaic;

extern "C" void launch(void* __raw_A, void* __raw_B, void* __raw_C) {
    using T = bfloat16_t;
    using MNKPSlabShape = decltype(static_tree_cast<int64_t>(Shape<Int<4096>, Int<64>, Int<2304>, Int<4>>{}));
    using MNKTileShape = decltype(static_tree_cast<int64_t>(Shape<Int<128>, Int<16>, Int<64>>{}));
    constexpr int d = 64, d_tile = 8;
    constexpr int pow = 2;
    constexpr bool expand_K = true;
    using GaSlab = decltype(static_tree_cast<int64_t>(Layout<Shape<Int<4096>, Int<64>, Int<4>>, Stride<Int<64>, Int<1>, Int<262144>>>{}));
    using GBSlab = decltype(static_tree_cast<int64_t>(Layout<Shape<Int<64>, Int<2304>, Int<4>>, Stride<Int<1>, Int<64>, Int<147456>>>{}));
    using GCSlab = decltype(static_tree_cast<int64_t>(Layout<Shape<Int<4096>, Int<64>, Int<4>>, Stride<Int<64>, Int<1>, Int<262144>>>{}));
    constexpr bool duplicate_correction = true;
    using PerfMosaic = PerfMosaic<1, 1, true, 1>;
    using Atom = MMA_Atom<SM80_16x8x16_F32BF16BF16F32_TN>;
    using MNKAtomPlacement = decltype(static_tree_cast<int64_t>(Shape<Int<2>, Int<2>, Int<1>>{}));
    using Mosaic = SympowMmaMosaic<T, pow, Atom, MNKAtomPlacement, MNKPSlabShape, MNKTileShape, d, d_tile, expand_K, GaSlab, GBSlab, GCSlab, PerfMosaic>;
    auto A = reinterpret_cast<T*>(__raw_A);
    auto B = reinterpret_cast<T*>(__raw_B);
    auto C = reinterpret_cast<T*>(__raw_C);
    launch_sympow_mma_kernel<duplicate_correction>(Mosaic{}, A, B, C);
}