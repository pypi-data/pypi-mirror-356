#include "hip/hip_runtime.h"
#include <Python.h>
#include <float.h>
// #include "./plavchan_periodogram.cu"

// START PLAVCHAN_PERIODOGRAM.CU
// IMPORT NOT INCLUDED, IT BREAKS PYPI BUILD

#include <float.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

static int nBlocks = 256;
static int nThreads = 512;

typedef struct {
    float* array;
    size_t dim1;
} Array1D;

typedef struct {
    float** array;
    size_t dim1;
    size_t* dim2;
} Array2D;

__device__ void logArr1D(Array1D* arr) {
    for (size_t i = 0; i < arr->dim1; i++) {
        printf("%f ", arr->array[i]);
    }
    printf("\n");
}

__device__ float getMax(Array1D* arr) {
    float max = -FLT_MAX;
    for (int i = 0; i < arr->dim1; i++) {
        if (arr->array[i] > max) {
            max = arr->array[i];
        }
    }
    return max;
}

__device__ float getMin(Array1D* arr) {
    float min = FLT_MAX;
    for (int i = 0; i < arr->dim1; i++) {
        if (arr->array[i] < min) {
            min = arr->array[i];
        }
    }
    return min;
}

__device__ void swap(float* a, float* b)
{
    float t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition(Array1D* arr, Array1D* sim, int l, int h)
{
    float x = arr->array[h];
    int i = (l - 1);
 
    for (int j = l; j <= h - 1; j++) {
        if (arr->array[j] <= x) {
            i++;
            swap(arr->array + i, arr->array + j);
            swap(sim->array + i, sim->array + j);
        }
    }
    swap(&arr->array[i + 1], &arr->array[h]);
    swap(&sim->array[i + 1], &sim->array[h]);
    return (i + 1);
}
 
__device__ void simulSort(Array1D* main, Array1D* sim, float* stack_buf) {   
    if (main->dim1 != sim->dim1) {
        printf("Error: Array dimensions do not match.\n");
        return;
    }

    // initialize top of stack
    int top = -1;
    int l = 0;
    int h = main->dim1 - 1;
 
    // push initial values of l and h to stack
    stack_buf[++top] = l;
    stack_buf[++top] = h;

    // Keep popping from stack while is not empty
    while (top >= 0) {
        // Pop h and l
        h = stack_buf[top--];
        l = stack_buf[top--];
 
        // Set pivot element at its correct position
        // in sorted array
        int p = partition(main, sim, (int)l, (int)h);
 
        // If there are elements on left side of pivot,
        // then push left side to stack
        if (p - 1 > l) {
            stack_buf[++top] = l;
            stack_buf[++top] = p - 1;
        }
 
        // If there are elements on right side of pivot,
        // then push right side to stack
        if (p + 1 < h) {
            stack_buf[++top] = p + 1;
            stack_buf[++top] = h;
        }
    }
}

__device__ void foldLC(Array1D* mag, Array1D* time, float modulus, float* stack_buf) {
    // raise error if modulus is 0
    if (modulus == 0) {
        printf("Error: Modulus cannot be zero.\n");

        for (int i = 0; i < time->dim1; i++) {
            time->array[i] = 0.0;
        }
    }

    // fold the light curve
    for (int i = 0; i < time->dim1; i++) {
        time->array[i] = fmodf(time->array[i], modulus); // time array should already be zeroed
        time->array[i] /= modulus; // normalize to [0,1]
    }

    simulSort(time, mag, stack_buf);
}

__device__ void boxcar_smoothing(Array1D* m, Array1D* t, float width, Array1D* smoothed) {
    /*
    Writes into `smoothed` a boxcar-smoothed equal-length version of magnitudes
    m: value array
    t: timestamp array on [0,1]
    */
    float halfWidth = width / 2;
    size_t N = t->dim1;
    float runningSum = 0;

    size_t l = 0; // left pointer increment
    size_t r = 0; // right pointer increment    

    for (size_t i = 0; i < N; i++) { // Generate a smoothed value for each point in t
        float leftLimitValue = t->array[i] - halfWidth;
        float rightLimitValue = t->array[i] + halfWidth;

        while (l < r && t->array[l] < leftLimitValue) {
            runningSum -= m->array[l];
            l++;
        }

        while (r < N && t->array[r] < rightLimitValue) {
            runningSum += m->array[r];
            r++;
        }

        smoothed->array[i] = runningSum / (r - l + 1); // average
    }
}

__device__ float plavchan_metric(Array1D* mag, Array1D* time, float width, Array1D* smoothed) { 
    boxcar_smoothing(mag, time, width, smoothed);

    float residue = 0;
    for (size_t i = 0; i < mag->dim1; i++) {
        float raw_diff = mag->array[i] - smoothed->array[i];
        residue += raw_diff * raw_diff;
    }

    return residue;
}

__global__ void plavchan_kernel(Array2D* mags, Array2D* times, Array1D* periods, float* width, 
    Array2D* periodogram, int objId, Array2D* folded_mags_buf , Array2D* folded_times_buf, Array2D* smoothed_buf) {
    
    /*
    mags: array of arrays of magnitudes
    time: array of arrays of times, same size as mags
    periods: array of trial periods
    width: fractional width of the boxcar smoothing, between 0 and 1
    periodogram: array of arrays of period values
    objId: the object ID we are working on
    folded_*_buf: buffers for the folded light curves. Dimension (n_concurrent_threads, max_lc_length)
    */

    int periodsPerThread = periods->dim1 / (gridDim.x * blockDim.x) + 1;
    int tId = blockIdx.x * blockDim.x + threadIdx.x;
    int startIdx = tId * periodsPerThread;
    int endIdx = startIdx + periodsPerThread;
    
    if (endIdx > periods->dim1) {
        endIdx = periods->dim1;
    }

    if (startIdx >= periods->dim1) {
        return;
    }

    size_t N = mags->dim2[objId]; // number of points in the light curve, equal to the number of times

    for (size_t i = startIdx; i < endIdx; i++) {
        float period = periods->array[i];

        Array1D folded_mag;
        Array1D folded_time;
        Array1D smoothed;

        folded_mag.array = folded_mags_buf->array[tId];
        folded_time.array = folded_times_buf->array[tId];
        smoothed.array = smoothed_buf->array[tId];
        folded_mag.dim1 = N;
        folded_time.dim1 = N;
        smoothed.dim1 = N;
        
        // copy over the light curve
        for (size_t j = 0; j < N; j++) {
            folded_mag.array[j] = mags->array[objId][j];
            folded_time.array[j] = times->array[objId][j];
        }

        foldLC(&folded_mag, &folded_time, period, smoothed.array); // fold the light curve, uses smooth buffer as stack space for sorting

        float score = plavchan_metric(&folded_mag, &folded_time, *width, &smoothed); // calculate the score
        periodogram->array[objId][i] = N / score; // store the score in the periodogram
    }

    return;
}

// void logArr2D(Array2D* arr) {
//     for (size_t i = 0; i < arr->dim1; i++) {
//         for (size_t j = 0; j < arr->dim2[i]; j++) {
//             printf("%f ", arr->array[i][j]);
//         }
//         printf("\n");
//     }
// }

// void logArr1D(Array1D* arr) {
//     for (size_t i = 0; i < arr->dim1; i++) {
//         printf("%f ", arr->array[i]);
//     }
//     printf("\n");
// }

static Array2D plavchan_periodogram(Array2D mags, Array2D times, Array1D pds, float width, int nBlocksPy, int nThreadsPy) {

    nBlocks = max(1, nBlocksPy);
    nThreads = max(1, nThreadsPy);

    // auto start =  std::chrono::high_resolution_clock::now();

    size_t max_len = 0;
    for (size_t i = 0; i < mags.dim1; i++) {
        max_len = max(max_len, mags.dim2[i]);
    }

    // Allocate and copy MAGS to the GPU
    Array2D* d_mags;
    hipMalloc(&d_mags, sizeof(Array2D));
    hipMemcpy(d_mags, &mags, sizeof(Array2D), hipMemcpyHostToDevice);

    size_t* d_mags_dim2;
    hipMalloc(&d_mags_dim2, mags.dim1 * sizeof(size_t));
    hipMemcpy(d_mags_dim2, mags.dim2, mags.dim1 * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_mags->dim2), &d_mags_dim2, sizeof(size_t*), hipMemcpyHostToDevice);

    float** d_mags_array;
    hipMalloc(&d_mags_array, mags.dim1 * sizeof(float*));
    for (size_t i = 0; i < mags.dim1; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, mags.dim2[i] * sizeof(float));
        hipMemcpy(d_tempRow, mags.array[i], mags.dim2[i] * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_mags_array + i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_mags->array), &d_mags_array, sizeof(float**), hipMemcpyHostToDevice);

    // Allocate and copy TIMES to the GPU
    Array2D* d_times;
    hipMalloc(&d_times, sizeof(Array2D));
    hipMemcpy(d_times, &times, sizeof(Array2D), hipMemcpyHostToDevice);

    size_t* d_times_dim2;
    hipMalloc(&d_times_dim2, times.dim1 * sizeof(size_t));
    hipMemcpy(d_times_dim2, times.dim2, times.dim1 * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_times->dim2), &d_times_dim2, sizeof(size_t*), hipMemcpyHostToDevice);

    float** d_times_array;
    hipMalloc(&d_times_array, times.dim1 * sizeof(float*));
    for (size_t i = 0; i < times.dim1; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, times.dim2[i] * sizeof(float));
        hipMemcpy(d_tempRow, times.array[i], times.dim2[i] * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_times_array + i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_times->array), &d_times_array, sizeof(float**), hipMemcpyHostToDevice);

    // Allocate and copy TRIAL PERIODS to the GPU
    Array1D* d_periods;
    hipMalloc(&d_periods, sizeof(Array1D));
    hipMemcpy(d_periods, &pds, sizeof(Array1D), hipMemcpyHostToDevice);

    float* d_periods_array;
    hipMalloc(&d_periods_array, pds.dim1 * sizeof(float));
    hipMemcpy(d_periods_array, pds.array, pds.dim1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&(d_periods->array), &d_periods_array, sizeof(float*), hipMemcpyHostToDevice);

    // Allocate and copy WIDTH to the GPU
    float* d_width;
    hipMalloc(&d_width, sizeof(float));
    hipMemcpy(d_width, &width, sizeof(float), hipMemcpyHostToDevice);

    // Create the output variable PERIODGRAM
    Array2D periodogram;
    periodogram.dim1 = mags.dim1;
    periodogram.dim2 = (size_t*)malloc(mags.dim1 * sizeof(size_t));
    for (size_t i = 0; i < mags.dim1; i++) {
        periodogram.dim2[i] = pds.dim1;
    }
    periodogram.array = (float**)malloc(mags.dim1 * sizeof(float*));
    for (size_t i = 0; i < mags.dim1; i++) {
        periodogram.array[i] = (float*)calloc(pds.dim1, sizeof(float));
    }

    // Allocate and copy PERIODGRAM to the GPU
    Array2D* d_periodogram;
    hipMalloc(&d_periodogram, sizeof(Array2D));
    hipMemcpy(d_periodogram, &periodogram, sizeof(Array2D), hipMemcpyHostToDevice);

    size_t* d_periodogram_dim2;
    hipMalloc(&d_periodogram_dim2, periodogram.dim1 * sizeof(size_t));
    hipMemcpy(d_periodogram_dim2, periodogram.dim2, periodogram.dim1 * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_periodogram->dim2), &d_periodogram_dim2, sizeof(size_t*), hipMemcpyHostToDevice);

    float** d_periodogram_array;
    hipMalloc(&d_periodogram_array, periodogram.dim1 * sizeof(float*));
    for (size_t i = 0; i < periodogram.dim1; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, periodogram.dim2[i] * sizeof(float));
        hipMemcpy(d_tempRow, periodogram.array[i], periodogram.dim2[i] * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_periodogram_array + i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_periodogram->array), &d_periodogram_array, sizeof(float**), hipMemcpyHostToDevice);

    // Allocate buffers for folded light curves
    int n_concurrent_threads = nBlocks * nThreads;

    // MAGS BUF
    Array2D* d_folded_mags_buf;
    hipMalloc(&d_folded_mags_buf, sizeof(Array2D));

    float* d_folded_mags_buf_dim2;
    hipMalloc(&d_folded_mags_buf_dim2, n_concurrent_threads * sizeof(size_t));

    float** d_folded_mags_buf_array;
    hipMalloc(&d_folded_mags_buf_array, n_concurrent_threads*sizeof(float*));
    for (size_t i = 0; i < n_concurrent_threads; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, max_len*sizeof(float)); // Expanded to longest possible lightcurve
        hipMemcpy(d_folded_mags_buf_array+i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_folded_mags_buf_dim2 +i, &max_len, sizeof(size_t), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_folded_mags_buf->array), &d_folded_mags_buf_array, sizeof(float**), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_mags_buf->dim1), &n_concurrent_threads, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_mags_buf->dim2), &d_folded_mags_buf_dim2, sizeof(size_t*), hipMemcpyHostToDevice);


    // TIMES BUF
    Array2D* d_folded_times_buf;
    hipMalloc(&d_folded_times_buf, sizeof(Array2D));

    float* d_folded_times_buf_dim2;
    hipMalloc(&d_folded_times_buf_dim2, n_concurrent_threads * sizeof(size_t));

    float** d_folded_times_buf_array;
    hipMalloc(&d_folded_times_buf_array, n_concurrent_threads*sizeof(float*));
    for (size_t i = 0; i < n_concurrent_threads; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, max_len*sizeof(float)); // Expanded to longest possible lightcurve
        hipMemcpy(d_folded_times_buf_array+i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_folded_times_buf_dim2 +i, &max_len, sizeof(size_t), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_folded_times_buf->array), &d_folded_times_buf_array, sizeof(float**), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_times_buf->dim1), &n_concurrent_threads, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_times_buf->dim2), &d_folded_times_buf_dim2, sizeof(size_t*), hipMemcpyHostToDevice);

    // SMOOTHED BUF
    Array2D* d_folded_smoothed_buf;
    hipMalloc(&d_folded_smoothed_buf, sizeof(Array2D));

    float* d_folded_smoothed_buf_dim2;
    hipMalloc(&d_folded_smoothed_buf_dim2, n_concurrent_threads * sizeof(size_t));

    float** d_folded_smoothed_buf_array;
    hipMalloc(&d_folded_smoothed_buf_array, n_concurrent_threads*sizeof(float*));
    for (size_t i = 0; i < n_concurrent_threads; i++) {
        float* d_tempRow;
        hipMalloc(&d_tempRow, max_len*sizeof(float)); // Expanded to longest possible lightcurve
        hipMemcpy(d_folded_smoothed_buf_array+i, &d_tempRow, sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_folded_smoothed_buf_dim2 +i, &max_len, sizeof(size_t), hipMemcpyHostToDevice);
    }
    hipMemcpy(&(d_folded_smoothed_buf->array), &d_folded_smoothed_buf_array, sizeof(float**), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_smoothed_buf->dim1), &n_concurrent_threads, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&(d_folded_smoothed_buf->dim2), &d_folded_smoothed_buf_dim2, sizeof(size_t*), hipMemcpyHostToDevice);

    // auto end = std::chrono::high_resolution_clock::now();

    // printf("Time taken for memory allocation: %lld ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
    // fflush(stdout);
    // Launch Kernel
    
    // printf("Launching kernel with %d blocks and %d threads on %d periods \n", nBlocks, nThreads, pds.dim1);

    // size_t freeMem, totalMem;
    // hipMemGetInfo(&freeMem, &totalMem);
    // freeMem /= 1024*1024; // convert to MB
    // totalMem /= 1024*1024; // convert to MB
    // printf("Free memory: %zu/%zu MB\n", freeMem, totalMem);
    // fflush(stdout);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error before kernel execution: %s\n", hipGetErrorString(err));
    }

    // start = std::chrono::high_resolution_clock::now();
    for (size_t objId = 0; objId < mags.dim1; objId++) {
        plavchan_kernel<<<nBlocks, nThreads>>>(d_mags, d_times, d_periods, d_width, d_periodogram, objId, 
            d_folded_mags_buf, d_folded_times_buf, d_folded_smoothed_buf);
    }

    err = hipDeviceSynchronize();

    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error during kernel execution: %s\n", hipGetErrorString(err));
    }
    // end = std::chrono::high_resolution_clock::now();
    // printf("Kernel execution finished\n");
    // printf("Time taken for computation: %lld ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
    // fflush(stdout);

    // Copy PERIODGRAM back to host
    for (size_t i = 0; i < periodogram.dim1; i++) {
        float* h_tempRow;
        hipMemcpy(&h_tempRow, d_periodogram_array + i, sizeof(float*), hipMemcpyDeviceToHost);
        hipMemcpy(periodogram.array[i], h_tempRow, periodogram.dim2[i] * sizeof(float), hipMemcpyDeviceToHost);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error during copying: %s\n", hipGetErrorString(err));
    }

    // Free GPU memory
    hipDeviceReset(); 

    // Z score the periodogram

    for (size_t i = 0; i < periodogram.dim1; i++) {
        float mean = 0;
        float stddev = 0;
        for (size_t j = 0; j < periodogram.dim2[i]; j++) {
            mean += periodogram.array[i][j];
        }
        mean /= periodogram.dim2[i];

        for (size_t j = 0; j < periodogram.dim2[i]; j++) {
            stddev += (periodogram.array[i][j] - mean) * (periodogram.array[i][j] - mean);
        }
        stddev = sqrt(stddev / periodogram.dim2[i]);

        for (size_t j = 0; j < periodogram.dim2[i]; j++) {
            periodogram.array[i][j] = (periodogram.array[i][j] - mean) / stddev;
        }
    }

    // return the proper object
    return periodogram;
}

// START PLAVCHAN.CU

void except(const char* str) { // Throws python exception
    PyErr_SetString(PyExc_TypeError, str);
    exit(1);
}

Array1D parseList(PyObject* list) {
    if (!PyList_Check(list)) {
        except("ERROR: Input must be a list.");
    } 

    Py_ssize_t n_entries = PyList_Size(list);
    float* c_arr = (float*)malloc(n_entries * sizeof(float));
    for (Py_ssize_t i = 0; i < n_entries; i++) {
        PyObject* entry = PyList_GetItem(list, i);
        if (!PyFloat_Check(entry)) {
            except("Entries must be floats");
        }

        c_arr[i] = (float)PyFloat_AsDouble(entry);
    }
    
    Array1D returnval;
    returnval.array = c_arr;
    returnval.dim1 = n_entries;
    return returnval;
}

Array2D parseListofLists(PyObject* lists) {
    if (!PyList_Check(lists)) {
        except("ERROR: Input must be a list.");
    }

    Py_ssize_t n_rows = PyList_Size(lists);
    float** c_arrs = (float**)malloc(n_rows * sizeof(float*));
    size_t* dim2 = (size_t*)malloc(sizeof(size_t) * n_rows);

    for (Py_ssize_t i = 0; i < n_rows; i++) {
        PyObject* innerList = PyList_GetItem(lists, i);
        if (!PyList_Check(innerList)) {
            except("ERROR: Each item in outer list must be a list.");
        }

        Py_ssize_t n_entries = PyList_Size(innerList);
        *(c_arrs + i) = (float*)malloc(n_entries * sizeof(float));
        *(dim2 + i) = n_entries;

        for (Py_ssize_t j = 0; j < n_entries; j++) {
            PyObject* entry = PyList_GetItem(innerList, j);
            if (!PyFloat_Check(entry)) {
                except("Entries must be floats");
            }

            float c_float_entry = (float)PyFloat_AsDouble(entry);
            *(*(c_arrs + i) + j) = c_float_entry;
        }
    }

    Array2D returnval;
    returnval.array = c_arrs;
    returnval.dim1 = n_rows;
    returnval.dim2 = dim2;
    return returnval;
}


static PyObject* PY_plavchan_periodogram(PyObject* self, PyObject* args) {
    PyObject* pymags;
    PyObject* pytimes;
    PyObject* pytrialperiods;
    float width;
    int device_id;
    int nBlocksPy;
    int nThreadsPy;
    if (PyArg_ParseTuple(args, "OOOfiii", &pymags, &pytimes, &pytrialperiods, &width, &nBlocksPy, &nThreadsPy, &device_id) == 0) {
        return NULL;
    }

    // Parse Python objects into C structures
    Array2D mags = parseListofLists(pymags);
    Array2D times = parseListofLists(pytimes);
    Array1D pds = parseList(pytrialperiods);

    // Safety checks
    if (mags.dim1 != times.dim1) {
        except("Mags and times mismatch in object count.");
        return NULL;
    }
    for (size_t i = 0; i < mags.dim1; i++) {
        if (mags.dim2[i] != times.dim2[i]) {
            char error_message[100];
            snprintf(error_message, sizeof(error_message), 
                     "Mags and times mismatch in entry count in object %zu.", i);
            except(error_message);
            return NULL;
        }
    }


    hipError_t device_error = hipSetDevice(device_id);
    if (device_error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(device_error));
        Py_RETURN_NONE;
    }

    Array2D periodogram = plavchan_periodogram(mags, times, pds, width, nBlocksPy, nThreadsPy); // actual working step

    // Convert PERIODGRAM to Python object
    PyObject* py_periodogram = PyList_New(periodogram.dim1);
    for (size_t i = 0; i < periodogram.dim1; i++) {
        PyObject* py_tempRow = PyList_New(periodogram.dim2[i]);
        for (size_t j = 0; j < periodogram.dim2[i]; j++) {
            PyObject* py_value = PyFloat_FromDouble(periodogram.array[i][j]);
            PyList_SetItem(py_tempRow, j, py_value);
        }
        PyList_SetItem(py_periodogram, i, py_tempRow);
    }

    return py_periodogram;
}

static PyObject* PY_get_device_count(PyObject* self, PyObject* args) {
    int device_count;
    hipError_t device_error = hipGetDeviceCount(&device_count);
    if (device_error != hipSuccess) {
        printf("CUDA error: %s", hipGetErrorString(device_error));
        Py_RETURN_NONE;
    }
    return PyLong_FromLong(device_count);
}

// Python integration stuff
static struct PyMethodDef methods[] = {
    {"__cuda__plavchan_pgram", (PyCFunction)PY_plavchan_periodogram, METH_VARARGS, "Compute Plavchan periodogram on GPU"}, 
    {"get_device_count", (PyCFunction)PY_get_device_count, METH_NOARGS, "Get number of CUDA devices"},
    {NULL, NULL, 0, NULL} 
};

static struct PyModuleDef module = {
    PyModuleDef_HEAD_INIT,
    "plavchan", 
    NULL,
    -1,
    methods
};

PyMODINIT_FUNC PyInit_plavchan(void) { 
    return PyModule_Create(&module);
}
