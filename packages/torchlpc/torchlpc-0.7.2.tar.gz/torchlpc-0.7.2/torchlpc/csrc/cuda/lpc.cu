#include "hip/hip_runtime.h"
#include <assert.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAGuard.h>
#include <stdio.h>
#include <torch/script.h>
#include <torch/torch.h>

// CUDA kernel for LPC computation
template <typename scalar_t>
__global__ void lpc_cuda_kernel(scalar_t* padded_y,  // [B, T + order]
                                const scalar_t* A,   // [B, T, order]
                                int64_t B, int64_t T, int64_t order) {
    extern __shared__ char smem[];
    scalar_t* sm = reinterpret_cast<scalar_t*>(smem);

    int b = blockIdx.x;
    int i = threadIdx.x;

    if (b >= B || i >= order) return;

    // Initialize shared memory with the first 'order' elements
    sm[i] = padded_y[b * (T + order) + i];
    __syncthreads();

    int circular_idx = 0;
    for (int t = 0; t < T; ++t) {
        circular_idx = t % order;
        scalar_t a = -A[((b * T + t) * order) + i];

        // Compute s as in the Python code
        int idx_offset = circular_idx - i - 1;
        if (i > circular_idx - 1) {
            idx_offset += order;
        }
        scalar_t s = sm[(idx_offset + order) % order];

        scalar_t v = a * s;

        if (i == order - 1) {
            sm[circular_idx] = v;
            v = padded_y[b * (T + order) + t + order];
        }
        __syncthreads();

        // Atomic add to shared memory
        atomicAdd(&sm[circular_idx], v);
        __syncthreads();

        if (i == order - 1) {
            padded_y[b * (T + order) + t + order] = sm[circular_idx];
        }
        __syncthreads();
    }
}
// CUDA kernel for complex LPC computation
template <typename scalar_t>
__global__ void lpc_cuda_kernel_complex(
    scalar_t* padded_y_real,  // [B, T + order]
    scalar_t* padded_y_imag,  // [B, T + order]
    const scalar_t* A_real,   // [B, T, order]
    const scalar_t* A_imag,   // [B, T, order]
    int64_t B, int64_t T, int64_t order) {
    extern __shared__ char smem[];
    scalar_t* sm_real = reinterpret_cast<scalar_t*>(smem);
    scalar_t* sm_imag = sm_real + order;

    int b = blockIdx.x;
    int i = threadIdx.x;

    if (b >= B || i >= order) return;

    // Initialize shared memory with the first 'order' elements
    sm_real[i] = padded_y_real[b * (T + order) + i];
    sm_imag[i] = padded_y_imag[b * (T + order) + i];
    __syncthreads();

    int circular_idx = 0;
    for (int t = 0; t < T; ++t) {
        circular_idx = t % order;
        scalar_t a_real = -A_real[((b * T + t) * order) + i];
        scalar_t a_imag = -A_imag[((b * T + t) * order) + i];

        int idx_offset = circular_idx - i - 1;
        if (i > circular_idx - 1) {
            idx_offset += order;
        }
        int s_idx = (idx_offset + order) % order;
        scalar_t s_real = sm_real[s_idx];
        scalar_t s_imag = sm_imag[s_idx];

        // Complex multiply: v = a * s
        scalar_t v_real = a_real * s_real - a_imag * s_imag;
        scalar_t v_imag = a_real * s_imag + a_imag * s_real;

        if (i == order - 1) {
            sm_real[circular_idx] = v_real;
            sm_imag[circular_idx] = v_imag;
            v_real = padded_y_real[b * (T + order) + t + order];
            v_imag = padded_y_imag[b * (T + order) + t + order];
        }
        __syncthreads();

        atomicAdd(&sm_real[circular_idx], v_real);
        atomicAdd(&sm_imag[circular_idx], v_imag);
        __syncthreads();

        if (i == order - 1) {
            padded_y_real[b * (T + order) + t + order] = sm_real[circular_idx];
            padded_y_imag[b * (T + order) + t + order] = sm_imag[circular_idx];
        }
        __syncthreads();
    }
}

at::Tensor lpc_cuda_wrapper(const at::Tensor& x, const at::Tensor& a,
                            const at::Tensor& zi) {
    TORCH_CHECK(x.is_floating_point() || x.is_complex(),
                "Input must be floating point or complex");
    TORCH_CHECK(a.scalar_type() == x.scalar_type(),
                "Coefficients must have the same scalar type as input");
    TORCH_CHECK(zi.scalar_type() == x.scalar_type(),
                "Initial conditions must have the same scalar type as input");

    TORCH_CHECK(x.dim() == 2, "Input must be 2D");
    TORCH_CHECK(zi.dim() == 2, "Initial conditions must be 2D");
    TORCH_CHECK(x.size(0) == zi.size(0),
                "Batch size of input and initial conditions must match");

    const at::cuda::OptionalCUDAGuard device_guard(device_of(x));

    auto a_contiguous = a.contiguous();

    at::Tensor out;
    auto order = a_contiguous.size(2);
    assert(order <= 1024 && "LPC order must be less than or equal to 1024");
    auto threads_per_block = order;

    if (x.is_floating_point()) {
        out = at::cat({zi.flip(1), x}, 1).contiguous();
        AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "lpc_cuda", [&] {
            auto padded_y = out.mutable_data_ptr<scalar_t>();
            auto A = a_contiguous.const_data_ptr<scalar_t>();
            auto B = x.size(0);
            auto T = x.size(1);

            lpc_cuda_kernel<scalar_t><<<B, threads_per_block,
                                        threads_per_block * sizeof(scalar_t)>>>(
                padded_y, A, B, T, order);
        });
    } else {
        auto out_real =
            at::cat({at::real(zi).flip(1), at::real(x)}, 1).contiguous();
        auto out_imag =
            at::cat({at::imag(zi).flip(1), at::imag(x)}, 1).contiguous();
        auto a_real = at::real(a_contiguous).contiguous();
        auto a_imag = at::imag(a_contiguous).contiguous();
        AT_DISPATCH_FLOATING_TYPES(
            out_real.scalar_type(), "lpc_cuda_complex", [&] {
                auto padded_y_real = out_real.mutable_data_ptr<scalar_t>();
                auto padded_y_imag = out_imag.mutable_data_ptr<scalar_t>();
                auto A_real = a_real.const_data_ptr<scalar_t>();
                auto A_imag = a_imag.const_data_ptr<scalar_t>();
                auto B = x.size(0);
                auto T = x.size(1);

                lpc_cuda_kernel_complex<scalar_t>
                    <<<B, threads_per_block,
                       2 * threads_per_block * sizeof(scalar_t)>>>(
                        padded_y_real, padded_y_imag, A_real, A_imag, B, T,
                        order);
            });
        out = at::view_as_complex(at::stack({out_real, out_imag}, -1));
    }
    return out.slice(1, order, out.size(1)).contiguous();
}

TORCH_LIBRARY_IMPL(torchlpc, CUDA, m) { m.impl("lpc", &lpc_cuda_wrapper); }